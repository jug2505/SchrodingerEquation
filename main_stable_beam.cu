#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <map>

using namespace std;

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)
template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        exit(1);
    }
}

#define SQRT_M_PI 1.77245385091

enum class Type{ FLEX, SOLID };

// Константы CUDA
#define BLOCK_SIZE 32

// Константы SPH
#define N 100
#define SOLID_LAYER_LENGTH 3
constexpr double DT = 0.02;  // Шаг по времени
constexpr int NT = 1500;  // Кол-во шагов по времени
constexpr int NT_SETUP = 0;  // Кол-во шагов на настройку
constexpr int N_OUT = 15;  // Вывод каждые N_OUT шагов
constexpr int N_PROGRESS = 10;
constexpr int PROGRESS_STEP = NT / N_PROGRESS;

// i d_t psi + nabla^2/2 psi -x^2 psi/2 = 0
// Потенциал: 1/2 x^2
double b = 0;  // Демпфирование скорости для настройки начального состояния
#define M (1.0 / N) // Масса частицы SPH ( M * n = 1 normalizes |wavefunction|^2 to 1)
#define H_DEFAULT (0.4)  // Расстояние сглаживания
#define H_COEF 8
constexpr double xStart = -3.0;
constexpr double xEnd = 3.0;
constexpr double xStep = (xEnd - xStart) / (N - 1);

// Коэффициенты задачи
double gamma0 = 4.32e-12;
double Kb = 1.38e-16;
double T = 77.0;
// chi=20 a=0.0065
// chi=40 a=0.0323
// chi=4 a=0.323
#define chi 20
double a_eq = 0.0065;
double b_eq = 2.0;
int m = 7;
#define ALPHA_MAX 9
#define L_MAX 9
double R = (-0.25*gamma0); // R = Q = -D = 0 , (-0.25*gamma0), (-0.5*gamma0)
double Q = R;
double D = -Q;

// Кол-во разбиений для интеграла
const int num_splits = 1000;

// Кэш
map<pair<int, int>, double> G_alpha_s_cache;
map<pair<int, int>, double> delta_alpha_s_cache;
map<int, double> G_alpha_cache;

// На GPU
double* x_dev;
double* xx_dev;
double* rho_dev;
double* drho_dev;
double* ddrho_dev;
double* P_dev;
double* u_dev;
double* a_dev;
double* mass_dev;
double* h_array_dev;
Type* particles_type_dev;
double* G_s_sum_array_dev;

// На CPU
double* x;
double* u;
double* rho;
double* drho;
double* ddrho;
double* P;
double* a;
double* xx; // Для графика
double* probe_rho; // Для графика
double* u_mhalf;
double* u_phalf;
double* mass;
double* h_array;
Type* particles_type;
double* G_s_sum_array;



void init() {
    x = new double[N];
    u = new double[N];
    rho = new double[N];
    drho = new double[N];
    ddrho = new double[N];
    P = new double[N];
    a = new double[N];
    xx = new double[N];
    probe_rho = new double[N];
    u_mhalf = new double[N];
    u_phalf = new double[N];
    mass = new double[N];
    h_array = new double[N];
    particles_type = new Type[N];
    G_s_sum_array = new double[ALPHA_MAX];


    hipMalloc(&x_dev, N * sizeof(double));
    hipMalloc(&xx_dev, N * sizeof(double));
    hipMalloc(&rho_dev, N * sizeof(double));
    hipMalloc(&drho_dev, N * sizeof(double));
    hipMalloc(&ddrho_dev, N * sizeof(double));
    hipMalloc(&P_dev, N * sizeof(double));
    hipMalloc(&u_dev, N * sizeof(double));
    hipMalloc(&a_dev, N * sizeof(double));
    hipMalloc(&mass_dev, N * sizeof(double));
    hipMalloc(&h_array_dev, N * sizeof(double));
    hipMalloc(&particles_type_dev, N * sizeof(Type));
    hipMalloc(&G_s_sum_array_dev, ALPHA_MAX * sizeof(double));
    checkCudaErrors(hipGetLastError());
}

void clear() {
    delete[] x;
    delete[] u;
    delete[] rho;
    delete[] drho;
    delete[] ddrho;
    delete[] P;
    delete[] a;
    delete[] xx;
    delete[] probe_rho;
    delete[] u_mhalf;
    delete[] u_phalf;
    delete[] mass;
    delete[] h_array;
    delete[] particles_type;
    delete[] G_s_sum_array;

    hipFree(x_dev);
    hipFree(xx_dev);
    hipFree(rho_dev);
    hipFree(drho_dev);
    hipFree(ddrho_dev);
    hipFree(P_dev);
    hipFree(u_dev);
    hipFree(a_dev);
    hipFree(mass_dev);
    hipFree(h_array_dev);
    hipFree(particles_type_dev);
    hipFree(G_s_sum_array_dev);
    checkCudaErrors(hipGetLastError());
}


__host__ __device__ double factorial(const int n) {
    double f = 1;
    for (int i=1; i<=n; ++i) f *= i;
    return f;
}

__host__ double F_func(const double p, const double s) {
    return 2.0 * gamma0 * D * (cos(2.0 * p / 3.0) + 2.0 * cos(p / 3.0) * cos(M_PI * s / m));
}

__host__ double eps(const double p, const double s) {
    return gamma0 * sqrt(1.0 + 4.0 * cos(p) * cos(M_PI * s / m) + 4.0 * cos(M_PI * s / m) * cos(M_PI * s / m));
}

__host__ double eps_imp(const double p, const double s) {
    return 0.5 * (R + Q + sqrt((R - Q) * (R - Q) - 4.0 * (F_func(p, s) - eps(p, s) * eps(p, s) - D*D)));
}

__host__ double deltaUnderIntegral(double p, double alpha, double s) {
    return eps_imp(p, s) * cos(p * alpha);
}

__host__ double simpsonIntegralDelta(const double a, const double b, const int n, double alpha, double s) {
    const double width = (b-a)/n;
    double simpson_integral = 0;
    for(int step = 0; step < n; step++) {
        const double x1 = a + step*width;
        const double x2 = a + (step+1)*width;
        simpson_integral += (x2-x1)/6.0*(deltaUnderIntegral(x1, alpha, s) + 4.0*deltaUnderIntegral(0.5*(x1+x2), alpha, s) + deltaUnderIntegral(x2, alpha, s));
    }
    return simpson_integral;
}

__host__ double delta(const int alpha, const int s) {
    if (delta_alpha_s_cache.find({alpha, s}) != delta_alpha_s_cache.end()) {
        return delta_alpha_s_cache[{alpha, s}];
    }
    double result = simpsonIntegralDelta(-M_PI, M_PI, num_splits, alpha, s) / M_PI;
    delta_alpha_s_cache[{alpha, s}] = result;
    cout << "delta_alpha_s alpha = " << alpha << ", s = " << s << " cached" << endl;
    return result;
}

__host__ double GNominatorUnderIntegral(double p, double alpha, double s) {
    double sum = delta(0, s) / (2.0 * Kb * T);
    for (double alpha = 1.0; alpha <= ALPHA_MAX; alpha++) {
        sum += delta(alpha, s) * cos(alpha * p) / (Kb * T);
    }
    return cos(alpha * p) / (1.0 + exp(sum));
}

__host__ double simpsonIntegralGNominator(const double a, const double b, const int n, double alpha, double s) {
    const double width = (b-a)/n;
    double simpson_integral = 0;
    for(int step = 0; step < n; step++) {
        const double x1 = a + step*width;
        const double x2 = a + (step+1)*width;
        simpson_integral += (x2-x1)/6.0*(GNominatorUnderIntegral(x1, alpha, s) + 4.0*GNominatorUnderIntegral(0.5*(x1+x2), alpha, s) + GNominatorUnderIntegral(x2, alpha, s));
    }
    return simpson_integral;
}

__host__ double simpsonIntegralGDenominator(double p, double alpha, double s) {
    double sum = delta(0, s) / (2.0 * Kb * T);
    for (double alpha = 1.0; alpha <= ALPHA_MAX; alpha++) {
        sum += delta(alpha, s) * cos(alpha * p) / (Kb * T);
    }
    return 1.0 / ( 1.0 + exp(sum));
}

__host__ double simpsonIntegralGDenominator(const double a, const double b, const int n, double alpha, double s) {
    const double width = (b-a)/n;
    double simpson_integral = 0;
    for(int step = 0; step < n; step++) {
        const double x1 = a + step*width;
        const double x2 = a + (step+1)*width;
        simpson_integral += (x2-x1)/6.0*(simpsonIntegralGDenominator(x1, alpha, s) + 4.0*simpsonIntegralGDenominator(0.5*(x1+x2), alpha, s) + simpsonIntegralGDenominator(x2, alpha, s));
    }
    return simpson_integral;
}

__host__ double G(const int alpha) {
    if (G_alpha_cache.find(alpha) != G_alpha_cache.end()) {
        return G_alpha_cache[alpha];
    }
    double nominator = 0.0;
    double denominator = 0.0;
    for(double s = 1.0; s <= m; s++) {
        nominator += delta(alpha, s) / gamma0 * simpsonIntegralGNominator(-M_PI, M_PI, num_splits, alpha, s);
        denominator += simpsonIntegralGDenominator(-M_PI, M_PI, num_splits, alpha, s);
    }
    double result = -alpha * (nominator / denominator);
    G_alpha_cache[alpha] = result;
    cout << "G_alpha alpha = " << alpha << " cached" << endl;
    return result;
}

__host__ __device__ double fl(double l) {
    return pow(-1, l) / (factorial(l) * pow(2, 2 * l) * tgamma(l + 2));
}

/* Гауссово сглаживающее ядро SPH (1D).
 * Вход: расстояния r, длина сглаживания h, порядок производной
 */
__device__ double kernelDeriv0(double r, double h) {
    return 1.0 / h / SQRT_M_PI * exp(- r * r / (h * h));
}

__device__ double kernelDeriv1(double r, double h) {
    return pow(h, -3) / SQRT_M_PI * exp(- r * r / (h * h)) * (-2.0 * r);
}

__device__ double kernelDeriv2(double r, double h) {
    return pow(h, -5) / SQRT_M_PI * exp(-r * r / (h * h)) * (4.0 * r * r - 2.0 * h * h);
}

__device__ double kernelDeriv3(double r, double h) {
    return pow(h, -7) / sqrt(M_PI) * exp(-pow(r, 2) / pow(h, 2)) * (-8.0 * pow(r, 3) + 12.0 * pow(h, 2) * r);
}

__global__ void densityKernel(double* x, double* mass, double* h_array, Type* particles_type, double* rho) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    if (particles_type[i] != Type::FLEX) return;

    double sum = 0.0;
    double x_i = x[i];
    double uij = 0.0;
    double hij = 0.0;
    for (int j = 0; j < N; j++) {
        uij = x_i - x[j];
        hij = (h_array[i] + h_array[j]) / 2.0;
        sum += mass[j] * kernelDeriv0(uij, hij);
    }
    rho[i] = sum;
    __syncthreads();
    h_array[i] = H_COEF * mass[i] / rho[i];

}

/* Вычисление плотности в каждом из мест расположения частиц с помощью сглаживающего ядра
 * Входные данные: позиции частиц x, масса SPH-частицы m, длина сглаживания h
 */
__host__ void density(double* x, double* rho) {
    hipMemcpy(x_dev, x, N * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = BLOCK_SIZE;
    int gridSize = (N + blockSize - 1) / blockSize;

    densityKernel<<<gridSize, blockSize>>>(x_dev, mass_dev, h_array_dev, particles_type_dev, rho_dev);

    hipMemcpy(rho, rho_dev, N * sizeof(double), hipMemcpyDeviceToHost);
    checkCudaErrors(hipGetLastError());
}

__global__ void pressureKernelDRho(double* x, double* mass, double* h_array, Type* particles_type, double* drho) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    if (particles_type[i] != Type::FLEX) return;

    double sum = 0.0;
    double x_i = x[i];
    double uij = 0.0;
    double hij = 0.0;
    for (int j = 0; j < N; j++) {
        uij = x_i - x[j];
        hij = (h_array[i] + h_array[j]) / 2.0;
        sum += mass[j] * kernelDeriv1(uij, hij);
    }
    drho[i] = sum;
}

__global__ void pressureKernelDDRho(double* x, double* mass, double* h_array, Type* particles_type, double* ddrho) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    if (particles_type[i] != Type::FLEX) return;

    double sum = 0.0;
    double x_i = x[i];
    double uij = 0.0;
    double hij = 0.0;
    for (int j = 0; j < N; j++) {
        uij = x_i - x[j];
        hij = (h_array[i] + h_array[j]) / 2.0;
        sum += mass[j] * kernelDeriv2(uij, hij);
    }
    ddrho[i] = sum;
}

__global__ void pressureKernel(double* x, double* rho, double* drho, double* ddrho, double* mass, double* h_array, Type* particles_type, double* G_s_sum_array, double* P) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    if (particles_type[i] != Type::FLEX) return;

    double sum = 0.0;
    double x_i = x[i];
    double uij = 0.0;
    double hij = 0.0;
    for (int j = 0; j < N; j++) {
        uij = x_i - x[j];
        hij = (h_array[i] + h_array[j]) / 2.0;
        sum += 0.25 * (drho[j] * drho[j] / rho[j] - ddrho[j]) / (chi * chi);
        sum = sum * mass[j] / rho[j] * kernelDeriv0(uij, hij);
    }
    P[i] = sum;
}

/* Вычисление давления на каждой из частиц с помощью сглаживающего ядра
 * P = -(1/4)*(d^2 rho /dx^2 - (d rho / dx)^2/rho)
 * Вход: положения x, плотности rho, масса SPH-частицы m, длина сглаживания h
 */
__host__ void pressure(double* x, double* rho, double* P) {
    hipMemcpy(x_dev, x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(rho_dev, rho, N * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = BLOCK_SIZE;
    int gridSize = (N + blockSize - 1) / blockSize;

    pressureKernelDRho<<<gridSize, blockSize>>>(x_dev, mass_dev, h_array_dev, particles_type_dev, drho_dev);
    pressureKernelDDRho<<<gridSize, blockSize>>>(x_dev, mass_dev, h_array_dev, particles_type_dev, ddrho_dev);

    hipMemcpy(drho, drho_dev, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(ddrho, ddrho_dev, N * sizeof(double), hipMemcpyDeviceToHost);

    pressureKernel<<<gridSize, blockSize>>>(x_dev, rho_dev, drho_dev, ddrho_dev, mass_dev, h_array_dev, particles_type_dev, G_s_sum_array_dev, P_dev);
    hipMemcpy(P, P_dev, N * sizeof(double), hipMemcpyDeviceToHost);
    checkCudaErrors(hipGetLastError());
}

__global__ void accelerationKernel(double* x, double* u, double* rho, double* P, double b, double* mass, double* h_array, Type* particles_type, double* G_s_sum_array, double* a) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    if (particles_type[i] != Type::FLEX) return;

    double sum = 0.0;
    double x_i = x[i];
    double uij = 0.0;
    double hij = 0.0;

    // Дэмпирование и гармонический потенциал (0.5 x^2)
//    a[i] = - u[i] * b - x[i];

    double sum_nl = 0.0;
    for (int alpha = 1; alpha <= ALPHA_MAX; alpha++) {
        double l_sum = 0.0;
        for (int l = 0; l <= L_MAX; l++) {
            l_sum += fl(l) * l / (l + 1.0) * pow(alpha, 2 * l + 1) * pow(rho[i], l + 1);
        }
        sum_nl += alpha * G_s_sum_array[alpha - 1] * l_sum;
    }
//        printf("%lf = %lf\n", sum, sum_nl);
    double P_NL = 1.0 / (2.0 * chi * chi) * sum_nl;

    for (int j = 0; j < N; j++) {
        uij = x_i - x[j];
        hij = (h_array[i] + h_array[j]) / 2.0;
        sum += -mass[j] * (P_NL / pow(rho[i], 2) + P[j] / (rho[j] * rho[j]) /*- P_NL/(rho[i] * rho[i])*/) * kernelDeriv1(uij, hij);
    }
    a[i] = sum;
}

/* Расчёт ускорения каждой частицы под действием квантового давления, гармонического потенциала, демпфирования скорости
 * Входные данные: положения x, скорости u, масса SPH-частицы m, плотность rho, давление P, коэффициент демпфирования b
 */
__host__ void acceleration(double* x, double* u, double* rho, double* P, double b, double* a) {
    hipMemcpy(x_dev, x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(rho_dev, rho, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(u_dev, u, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(P_dev, P, N * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = BLOCK_SIZE;
    int gridSize = (N + blockSize - 1) / blockSize;

    accelerationKernel<<<gridSize, blockSize>>>(x_dev, u_dev, rho_dev, P_dev, b, mass_dev, h_array_dev, particles_type_dev, G_s_sum_array_dev, a_dev);
    hipMemcpy(a, a_dev, N * sizeof(double), hipMemcpyDeviceToHost);
    checkCudaErrors(hipGetLastError());
}

__global__ void probeDensityKernel(double* x, double* xx, double* mass, double* h_array, double* rho) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    double sum = 0.0;
    double xx_i = xx[i];
    double uij = 0.0;
    double hij = 0.0;
    for (int j = 0; j < N; j++) {
        uij = xx_i - x[j];
        hij = (H_DEFAULT + h_array[j]) / 2.0;
        sum += mass[j] * kernelDeriv0(uij, hij);
    }
    rho[i] = sum;
}

/* Вычисление плотности в произвольных точках
 * Вход: положение x, масса частицы SPH m, масштабная длина h, точки измерения xx
 * Выход: плотность в равномерно расположенных точках
 */
__host__ void probeDensity(double* x, double* xx, double* prob_rho) {
    hipMemcpy(x_dev, x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(xx_dev, xx, N * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = BLOCK_SIZE;
    int gridSize = (N + blockSize - 1) / blockSize;

    probeDensityKernel<<<gridSize, blockSize>>>(x_dev, xx_dev, mass_dev, h_array_dev, rho_dev);

    hipMemcpy(prob_rho, rho_dev, N * sizeof(double), hipMemcpyDeviceToHost);
    checkCudaErrors(hipGetLastError());
}

void getCudaInfo() {
    int nDevices;
    hipGetDeviceCount(&nDevices);

    printf("Number of devices: %d\n", nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (MHz): %d\n",
               prop.memoryClockRate/1024);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
        printf("  Shared memory per block (Kbytes) %.1f\n",(float)(prop.sharedMemPerBlock)/1024.0);
        printf("  minor-major: %d-%d\n", prop.minor, prop.major);
        printf("  Warp-size: %d\n", prop.warpSize);
        printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n\n",prop.deviceOverlap ? "yes" : "no");
    }
    hipSetDevice(0);
}


void compute() {
    getCudaInfo();
    init();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Инициализация положений и скоростей частиц
    for (int i = 0; i < N; i++) {
        x[i] = xStart + i * xStep;
        xx[i] = x[i]; // Для графика
    }

    double v0 = (xStart + xEnd) / 2.0;
    for (int i = 0; i < N; i++) {
        if (i < SOLID_LAYER_LENGTH || i >= N - SOLID_LAYER_LENGTH) {
            rho[i] = a_eq;
            particles_type[i] = Type::SOLID;
        } else {
            rho[i] = a_eq * exp(-(x[i] - v0) * (x[i] - v0) / (b_eq * b_eq));
            particles_type[i] = Type::FLEX;
        }
    }
    hipMemcpy(rho_dev, rho, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(particles_type_dev, particles_type, N * sizeof(Type), hipMemcpyHostToDevice);

    // Инициализация масс частиц
    for (int i = 0; i < N; i++) {
        mass[i] = xStep * rho[i];//a_eq*a_eq * exp(-2.0 * (x[i] - v0) * (x[i] - v0) / (b_eq * b_eq));//xStep * rho[i];
    }
    hipMemcpy(mass_dev, mass, N * sizeof(double), hipMemcpyHostToDevice);

    // Инициализация сглаживающего расстояния
    for (int i = 0; i < N; i++) {
        h_array[i] = H_COEF * mass[i] / rho[i];
    }
    hipMemcpy(h_array_dev, h_array, N * sizeof(double), hipMemcpyHostToDevice);

    // Вычисление G_alpha
    double a0 = 0.0;
    double a1 = 0.0;
    for (int alpha = 1; alpha <= ALPHA_MAX; alpha++) {
        double G_alpha = G(alpha);
        G_s_sum_array[alpha - 1] = G_alpha;
        a0 += G_alpha * alpha;
        a1 -= G_alpha * alpha * alpha * alpha / 8.0;
    }
    hipMemcpy(G_s_sum_array_dev, G_s_sum_array, ALPHA_MAX * sizeof(double), hipMemcpyHostToDevice);
    cout << "SPH a0 = " << a0 << endl;
    cout << "SPH a1 = " << a1 << endl;


    // Инициализация плотности, давления и ускорения
    density(x, rho);
    pressure(x, rho, P);
    acceleration(x, u, rho, P, b, a);

    // v в t=-0.5*DT для leap frog интегратора
    for (int i = 0; i < N; i++) {
        u_mhalf[i] = u[i] - 0.5 * DT * a[i];
    }

    ofstream outfile("solution_cuda.txt");
    outfile << "X T Z" << endl;

    ofstream outfile_exact("solution_exact_cuda.txt");
    outfile_exact << "X T Z" << endl;

    // Главный цикл по времени
    double t = 0.0;
    for (int i = -NT_SETUP; i < NT; i++) {
        // Leap frog
        for (int j = 0; j < N; j++) {
            if (particles_type[j] != Type::FLEX) continue;
//            printf("%lf\n", a[j]);
            u_phalf[j] = u_mhalf[j] + a[j] * DT;
            x[j] = x[j] + u_phalf[j] * DT;
            u[j] = 0.5 * (u_mhalf[j] + u_phalf[j]);
            u_mhalf[j] = u_phalf[j];
        }

        if (i >= 0) {
            t = t + DT;
        }
        
        if ((i % PROGRESS_STEP) == 0) {
            int progress = (i / PROGRESS_STEP) + 1;
            cout << "SPH Progress: " << progress << "/" << N_PROGRESS << endl;
        }

        if (i == -1) {
            for (int j = 0; j < N; j++) {
                u_mhalf[j] = 1.0;
            }
            b = 0;
        }

        // Обновление плотностей, давлений, ускорений
        density(x, rho);
        pressure(x, rho, P);
        acceleration(x, u, rho, P, b, a);

        // Вывод в файлы
        if (i >= 0 && i % N_OUT == 0) {
            probeDensity(x, xx, probe_rho);
            for (int j = 0; j < N; j++) {
                outfile << xx[j] << " " << t << " " << probe_rho[j] / a_eq << endl; // TODO
//                outfile << x[j] << " " << t << " " << rho[j] / a_eq << endl;
            }
            for (int j = 0; j < N; j++) {
                double exact = 1.0 / sqrt(M_PI) * exp(-(xx[j] - sin(t)) * (xx[j]- sin(t)) / 2.0) * exp(-(xx[j] - sin(t)) * (xx[j]- sin(t)) / 2.0);
                outfile_exact << xx[j] << " " << t << " " << exact << endl;
            }
        }
    }
    outfile.close();
    outfile_exact.close();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Время работы: %3.1f s\n", elapsedTime / 1000.0);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    clear();
}


int main() {
    compute();
    return 0;
}
