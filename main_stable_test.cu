#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <chrono>
#include <map>

using namespace std;

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)
template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        exit(1);
    }
}

#define SQRT_M_PI 1.77245385091

enum class Type{ FLEX, SOLID };

// Константы CUDA
#define BLOCK_SIZE 32

// Константы SPH
#define N 500
#define SOLID_LAYER_LENGTH 0
constexpr double DT = 0.02;  // Шаг по времени
constexpr int NT = 500;  // Кол-во шагов по времени
constexpr int NT_SETUP = 800;  // Кол-во шагов на настройку
constexpr int N_OUT = 1;  // Вывод каждые N_OUT шагов
constexpr int N_PROGRESS = 10;
constexpr int PROGRESS_STEP = NT / N_PROGRESS;

// i d_t psi + nabla^2/2 psi -x^2 psi/2 = 0
// Потенциал: 1/2 x^2
double b = 4;  // Демпфирование скорости для настройки начального состояния
#define M (1.0 / N) // Масса частицы SPH ( M * n = 1 normalizes |wavefunction|^2 to 1)
#define H_DEFAULT (0.4)  // Расстояние сглаживания
#define H_COEF 1.3
constexpr double xStart = -3.0;
constexpr double xEnd = 3.0;
constexpr double xStep = (xEnd - xStart) / (N - 1);

// На GPU
double* x_dev;
double* xx_dev;
double* rho_dev;
double* drho_dev;
double* ddrho_dev;
double* P_dev;
double* u_dev;
double* a_dev;
double* mass_dev;
double* h_array_dev;
Type* particles_type_dev;

// На CPU
double* x;
double* u;
double* rho;
double* drho;
double* ddrho;
double* P;
double* a;
double* xx; // Для графика
double* probe_rho; // Для графика
double* u_mhalf;
double* u_phalf;
double* mass;
double* h_array;
Type* particles_type;



void init() {
    x = new double[N];
    u = new double[N];
    rho = new double[N];
    drho = new double[N];
    ddrho = new double[N];
    P = new double[N];
    a = new double[N];
    xx = new double[N];
    probe_rho = new double[N];
    u_mhalf = new double[N];
    u_phalf = new double[N];
    mass = new double[N];
    h_array = new double[N];
    particles_type = new Type[N];


    hipMalloc(&x_dev, N * sizeof(double));
    hipMalloc(&xx_dev, N * sizeof(double));
    hipMalloc(&rho_dev, N * sizeof(double));
    hipMalloc(&drho_dev, N * sizeof(double));
    hipMalloc(&ddrho_dev, N * sizeof(double));
    hipMalloc(&P_dev, N * sizeof(double));
    hipMalloc(&u_dev, N * sizeof(double));
    hipMalloc(&a_dev, N * sizeof(double));
    hipMalloc(&mass_dev, N * sizeof(double));
    hipMalloc(&h_array_dev, N * sizeof(double));
    hipMalloc(&particles_type_dev, N * sizeof(Type));
    checkCudaErrors(hipGetLastError());
}

void clear() {
    delete[] x;
    delete[] u;
    delete[] rho;
    delete[] drho;
    delete[] ddrho;
    delete[] P;
    delete[] a;
    delete[] xx;
    delete[] probe_rho;
    delete[] u_mhalf;
    delete[] u_phalf;
    delete[] mass;
    delete[] h_array;
    delete[] particles_type;

    hipFree(x_dev);
    hipFree(xx_dev);
    hipFree(rho_dev);
    hipFree(drho_dev);
    hipFree(ddrho_dev);
    hipFree(P_dev);
    hipFree(u_dev);
    hipFree(a_dev);
    hipFree(mass_dev);
    hipFree(h_array_dev);
    hipFree(particles_type_dev);
    checkCudaErrors(hipGetLastError());
}

/* Гауссово сглаживающее ядро SPH (1D).
 * Вход: расстояния r, длина сглаживания h, порядок производной
 */
__device__ double kernelDeriv0(double r, double h) {
    return 1.0 / h / SQRT_M_PI * exp(- r * r / (h * h));
}

__device__ double kernelDeriv1(double r, double h) {
    return pow(h, -3) / SQRT_M_PI * exp(- r * r / (h * h)) * (-2.0 * r);
}

__device__ double kernelDeriv2(double r, double h) {
    return pow(h, -5) / SQRT_M_PI * exp(-r * r / (h * h)) * (4.0 * r * r - 2.0 * h * h);
}

__device__ double kernelDeriv3(double r, double h) {
    return pow(h, -7) / sqrt(M_PI) * exp(-pow(r, 2) / pow(h, 2)) * (-8.0 * pow(r, 3) + 12.0 * pow(h, 2) * r);
}

__global__ void densityKernel(double* x, double* mass, double* h_array, Type* particles_type, double* rho) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    if (particles_type[i] != Type::FLEX) return;

    double sum = 0.0;
    double x_i = x[i];
    double uij = 0.0;
    double hij = 0.0;
    for (int j = 0; j < N; j++) {
        uij = x_i - x[j];
        hij = (h_array[i] + h_array[j]) / 2.0;
        sum += mass[j] * kernelDeriv0(uij, hij);
    }
    rho[i] = sum;
//    __syncthreads();
//    h_array[i] = H_COEF * mass[i] / rho[i];

}

/* Вычисление плотности в каждом из мест расположения частиц с помощью сглаживающего ядра
 * Входные данные: позиции частиц x, масса SPH-частицы m, длина сглаживания h
 */
__host__ void density(double* x, double* rho) {
    hipMemcpy(x_dev, x, N * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = BLOCK_SIZE;
    int gridSize = (N + blockSize - 1) / blockSize;

    densityKernel<<<gridSize, blockSize>>>(x_dev, mass_dev, h_array_dev, particles_type_dev, rho_dev);

    hipMemcpy(rho, rho_dev, N * sizeof(double), hipMemcpyDeviceToHost);
    checkCudaErrors(hipGetLastError());
}

__global__ void pressureKernelDRho(double* x, double* mass, double* h_array, Type* particles_type, double* drho) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    if (particles_type[i] != Type::FLEX) return;

    double sum = 0.0;
    double x_i = x[i];
    double uij = 0.0;
    double hij = 0.0;
    for (int j = 0; j < N; j++) {
        uij = x_i - x[j];
        hij = (h_array[i] + h_array[j]) / 2.0;
        sum += mass[j] * kernelDeriv1(uij, hij);
    }
    drho[i] = sum;
}

__global__ void pressureKernelDDRho(double* x, double* mass, double* h_array, Type* particles_type, double* ddrho) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    if (particles_type[i] != Type::FLEX) return;

    double sum = 0.0;
    double x_i = x[i];
    double uij = 0.0;
    double hij = 0.0;
    for (int j = 0; j < N; j++) {
        uij = x_i - x[j];
        hij = (h_array[i] + h_array[j]) / 2.0;
        sum += mass[j] * kernelDeriv2(uij, hij);
    }
    ddrho[i] = sum;
}

__global__ void pressureKernel(double* x, double* rho, double* drho, double* ddrho, double* mass, double* h_array, Type* particles_type, double* P) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    if (particles_type[i] != Type::FLEX) return;

    double sum = 0.0;
    double x_i = x[i];
    double uij = 0.0;
    double hij = 0.0;
    for (int j = 0; j < N; j++) {
        uij = x_i - x[j];
        hij = (h_array[i] + h_array[j]) / 2.0;
        sum += 0.25 * (drho[j] * drho[j] / rho[j] - ddrho[j]) * mass[j] / rho[j] * kernelDeriv0(uij, hij);
    }
    P[i] = sum;
}

/* Вычисление давления на каждой из частиц с помощью сглаживающего ядра
 * P = -(1/4)*(d^2 rho /dx^2 - (d rho / dx)^2/rho)
 * Вход: положения x, плотности rho, масса SPH-частицы m, длина сглаживания h
 */
__host__ void pressure(double* x, double* rho, double* P) {
    hipMemcpy(x_dev, x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(rho_dev, rho, N * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = BLOCK_SIZE;
    int gridSize = (N + blockSize - 1) / blockSize;

    pressureKernelDRho<<<gridSize, blockSize>>>(x_dev, mass_dev, h_array_dev, particles_type_dev, drho_dev);
    pressureKernelDDRho<<<gridSize, blockSize>>>(x_dev, mass_dev, h_array_dev, particles_type_dev, ddrho_dev);

    hipMemcpy(drho, drho_dev, N * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(ddrho, ddrho_dev, N * sizeof(double), hipMemcpyDeviceToHost);

    pressureKernel<<<gridSize, blockSize>>>(x_dev, rho_dev, drho_dev, ddrho_dev, mass_dev, h_array_dev, particles_type_dev, P_dev);
    hipMemcpy(P, P_dev, N * sizeof(double), hipMemcpyDeviceToHost);
    checkCudaErrors(hipGetLastError());
}

__global__ void accelerationKernel(double* x, double* u, double* rho, double* P, double b, double* mass, double* h_array, Type* particles_type, double* a) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    if (particles_type[i] != Type::FLEX) return;

    double sum = 0.0;
    double x_i = x[i];
    double uij = 0.0;
    double hij = 0.0;

    // Дэмпирование и гармонический потенциал (0.5 x^2)
    a[i] = - u[i] * b - x[i];

    for (int j = 0; j < N; j++) {
        uij = x_i - x[j];
        hij = (h_array[i] + h_array[j]) / 2.0;
        sum += -mass[j] * (P[i] / pow(rho[i], 2) + P[j] / (rho[j] * rho[j])) * kernelDeriv1(uij, hij);
    }
    a[i] = a[i] + sum;
}

/* Расчёт ускорения каждой частицы под действием квантового давления, гармонического потенциала, демпфирования скорости
 * Входные данные: положения x, скорости u, масса SPH-частицы m, плотность rho, давление P, коэффициент демпфирования b
 */
__host__ void acceleration(double* x, double* u, double* rho, double* P, double b, double* a) {
    hipMemcpy(x_dev, x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(rho_dev, rho, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(u_dev, u, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(P_dev, P, N * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = BLOCK_SIZE;
    int gridSize = (N + blockSize - 1) / blockSize;

    accelerationKernel<<<gridSize, blockSize>>>(x_dev, u_dev, rho_dev, P_dev, b, mass_dev, h_array_dev, particles_type_dev, a_dev);
    hipMemcpy(a, a_dev, N * sizeof(double), hipMemcpyDeviceToHost);
    checkCudaErrors(hipGetLastError());
}

__global__ void probeDensityKernel(double* x, double* xx, double* mass, double* h_array, double* rho) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    double sum = 0.0;
    double xx_i = xx[i];
    double uij = 0.0;
    double hij = 0.0;
    for (int j = 0; j < N; j++) {
        uij = xx_i - x[j];
        hij = (H_DEFAULT + h_array[j]) / 2.0;
        sum += mass[j] * kernelDeriv0(uij, hij);
    }
    rho[i] = sum;
}

/* Вычисление плотности в произвольных точках
 * Вход: положение x, масса частицы SPH m, масштабная длина h, точки измерения xx
 * Выход: плотность в равномерно расположенных точках
 */
__host__ void probeDensity(double* x, double* xx, double* prob_rho) {
    hipMemcpy(x_dev, x, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(xx_dev, xx, N * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = BLOCK_SIZE;
    int gridSize = (N + blockSize - 1) / blockSize;

    probeDensityKernel<<<gridSize, blockSize>>>(x_dev, xx_dev, mass_dev, h_array_dev, rho_dev);

    hipMemcpy(prob_rho, rho_dev, N * sizeof(double), hipMemcpyDeviceToHost);
    checkCudaErrors(hipGetLastError());
}

void getCudaInfo() {
    int nDevices;
    hipGetDeviceCount(&nDevices);

    printf("Number of devices: %d\n", nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (MHz): %d\n",
               prop.memoryClockRate/1024);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %.1f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Total global memory (Gbytes) %.1f\n",(float)(prop.totalGlobalMem)/1024.0/1024.0/1024.0);
        printf("  Shared memory per block (Kbytes) %.1f\n",(float)(prop.sharedMemPerBlock)/1024.0);
        printf("  minor-major: %d-%d\n", prop.minor, prop.major);
        printf("  Warp-size: %d\n", prop.warpSize);
        printf("  Concurrent kernels: %s\n", prop.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n\n",prop.deviceOverlap ? "yes" : "no");
    }
    hipSetDevice(0);
}


void compute() {
    getCudaInfo();
    init();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Инициализация положений и скоростей частиц
    for (int i = 0; i < N; i++) {
        x[i] = xStart + i * xStep;
        xx[i] = x[i]; // Для графика
    }

    double v0 = (xStart + xEnd) / 2.0;
    for (int i = 0; i < N; i++) {
        if (i < SOLID_LAYER_LENGTH || i >= N - SOLID_LAYER_LENGTH) {
            rho[i] = 0.3;
            particles_type[i] = Type::SOLID;
        } else {
//            rho[i] = 0.3 * exp(-(x[i] - v0) * (x[i] - v0) / (4.0));
            particles_type[i] = Type::FLEX;
        }
    }
    hipMemcpy(rho_dev, rho, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(particles_type_dev, particles_type, N * sizeof(Type), hipMemcpyHostToDevice);

    // Инициализация масс частиц
    for (int i = 0; i < N; i++) {
        mass[i] = M;//xStep * rho[i];//M;
    }
    hipMemcpy(mass_dev, mass, N * sizeof(double), hipMemcpyHostToDevice);

    // Инициализация сглаживающего расстояния
    for (int i = 0; i < N; i++) {
        h_array[i] = H_DEFAULT;//10 * mass[i] / rho[i];
    }
    hipMemcpy(h_array_dev, h_array, N * sizeof(double), hipMemcpyHostToDevice);

    // Инициализация плотности, давления и ускорения
    density(x, rho);
    pressure(x, rho, P);
    acceleration(x, u, rho, P, b, a);

    // v в t=-0.5*DT для leap frog интегратора
    for (int i = 0; i < N; i++) {
        u_mhalf[i] = u[i] - 0.5 * DT * a[i];
    }

    ofstream outfile("solution_cuda.txt");
    outfile << "X T Z" << endl;

    ofstream outfile_exact("solution_exact_cuda.txt");
    outfile_exact << "X T Z" << endl;

    ofstream outfile_rmse("solution_cuda_rmse.txt");
    outfile_rmse<< "T RMSE" << endl;

    // Главный цикл по времени
    double t = 0.0;
    for (int i = -NT_SETUP; i < NT; i++) {
        // Leap frog
        for (int j = 0; j < N; j++) {
            if (particles_type[j] != Type::FLEX) continue;
            u_phalf[j] = u_mhalf[j] + a[j] * DT;
            x[j] = x[j] + u_phalf[j] * DT;
            u[j] = 0.5 * (u_mhalf[j] + u_phalf[j]);
            u_mhalf[j] = u_phalf[j];
        }

        if (i >= 0) {
            t = t + DT;
        }
        
        if ((i % PROGRESS_STEP) == 0) {
            int progress = (i / PROGRESS_STEP) + 1;
            cout << "SPH Progress: " << progress << "/" << N_PROGRESS << endl;
        }

        if (i == -1) {
            for (int j = 0; j < N; j++) {
                u_mhalf[j] = 1.0;
            }
            b = 0;
        }

        // Обновление плотностей, давлений, ускорений
        density(x, rho);
        pressure(x, rho, P);
        acceleration(x, u, rho, P, b, a);

        // Вывод в файлы
        if (i >= 0 && i % N_OUT == 0) {
            probeDensity(x, xx, probe_rho);
            for (int j = 0; j < N; j++) {
                outfile << xx[j] << " " << t << " " << probe_rho[j] << endl; // TODO
//                outfile << x[j] << " " << t << " " << rho[j] << endl;
            }

            double rmse = 0.0;
            for (int j = 0; j < N; j++) {
                double exact = 1.0 / sqrt(M_PI) * exp(-(xx[j] - sin(t)) * (xx[j]- sin(t)) / 2.0) * exp(-(xx[j] - sin(t)) * (xx[j]- sin(t)) / 2.0);
                outfile_exact << xx[j] << " " << t << " " << exact << endl;

                rmse += (exact  - probe_rho[j]) * (exact  - probe_rho[j]);
            }
            rmse = sqrt(rmse / N);

            outfile_rmse << t << " " << rmse << endl;
        }
    }
    outfile.close();
    outfile_exact.close();
    outfile_rmse.close();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Время работы: %3.1f s\n", elapsedTime / 1000.0);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    clear();
}


int main() {
    compute();
    return 0;
}
